/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

void correlate(int ny, int nx, const float *data, float *result) {
    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, n * n * sizeof(float)));
    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}
